#include "hip/hip_runtime.h"
//-----------------------------------------------------------
//  Copyright (C) 2021 Piotr (Peter) Beben <pdbcas2@gmail.com>
//  See LICENSE included with this distribution.

// CUDA laplacian smoothing a surface mesh, adjusted for vertex normals
// to preserve surface curvature.

#include "stdafx.h"
#include "mesh_smooth.cuh"
#include "handle_error.cuh"
#include "devArray.cuh"

#include <array>
#include <vector>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//---------------------------------------------------------
// Shared memory implementation. Assumes mesh vertices are
// localized into patches, laid out in memory s.t. topologically 
// near vertices tend to be near in memory as well. Otherwise 
// blocking into shared memory won't improve performance. 
// More precisely, assumes that with high probability adjacent 
// vertices v and w satisfy v == verts[i] and w == verts[j] for
// some i and j such that |i-j| < blockDim.x.
//
// ** Note **: Assumes the subset of vertex indices vertidxs
//	to be smoothed is in increasing order and spaced densely, 
//  else again there might not be any performance benefit here.

__global__ void cuda_mesh_smooth_sharedmem(
	size_t nverts, size_t nidxs, 
	const size_t* vertidxs, const size_t *vertadj, 
	const size_t *vertadjOffsets,
	const float *vertsx, const float *vertsy, const float *vertsz,
	const float *normsx, const float *normsy, const float *normsz,
	float* newVertsx, float* newVertsy, float* newVertsz,
	float* newNormsx, float* newNormsy, float* newNormsz)
{
	extern __shared__ char buffer[];
	float3* vertsBlock = (float3*)&buffer[0];
	float3* normsBlock = (float3*)&buffer[blockDim.x*sizeof(float3)];

	const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const size_t lower = vertidxs[idx - threadIdx.x];
	const size_t upper = min(nverts, size_t(lower + blockDim.x)) - 1;

	size_t ivert = lower + threadIdx.x;

	if (ivert <= upper) {
		vertsBlock[threadIdx.x].x = vertsx[ivert];
		vertsBlock[threadIdx.x].y = vertsy[ivert];
		vertsBlock[threadIdx.x].z = vertsz[ivert];
		normsBlock[threadIdx.x].x = normsx[ivert];
		normsBlock[threadIdx.x].y = normsy[ivert];
		normsBlock[threadIdx.x].z = normsz[ivert];
	}
	__syncthreads();


	if (idx < nidxs) {
		ivert = vertidxs[idx];

		float3 v0, n0;
		if (ivert < lower || ivert > upper) {
			v0.x = vertsx[ivert];
			v0.y = vertsy[ivert];
			v0.z = vertsz[ivert];
			n0.x = normsx[ivert];
			n0.y = normsy[ivert];
			n0.z = normsz[ivert];
		}
		else {
			v0 = vertsBlock[ivert-lower];
			n0 = normsBlock[ivert-lower];
		}

		float3 v1 = make_float3(0.0f, 0.0f, 0.0f);
		float3 n1 = make_float3(0.0f, 0.0f, 0.0f);

		size_t offset = vertadjOffsets[idx];
		size_t offsetNxt = vertadjOffsets[idx+1];
		size_t size = offsetNxt - offset;
		for (size_t i = offset; i < offsetNxt; ++i) {
			size_t a = vertadj[i];

			float3 v, n;
			if (a < lower || a > upper) {
				v.x = vertsx[a];
				v.y = vertsy[a];
				v.z = vertsz[a];
				n.x = normsx[a];
				n.y = normsy[a];
				n.z = normsz[a];
			}
			else {
				v = vertsBlock[a-lower];
				n = normsBlock[a-lower];
			}

			v.x = v.x - v0.x;
			v.y = v.y - v0.y;
			v.z = v.z - v0.z;
			float vn0 = v.x*n0.x + v.y*n0.y + v.z*n0.z;
			float vn = v.x*n.x + v.y*n.y + v.z*n.z;
			v1.x += 0.5f*v.x + 0.25f*(vn*n.x - vn0*n0.x);
			v1.y += 0.5f*v.y + 0.25f*(vn*n.y - vn0*n0.y);
			v1.z += 0.5f*v.z + 0.25f*(vn*n.z - vn0*n0.z);

			n.x = 0.5f*(n.x + n0.x);
			n.y = 0.5f*(n.y + n0.y);
			n.z = 0.5f*(n.z + n0.z);
			float length = sqrt(n.x*n.x + n.y*n.y + n.z*n.z);	
			n1.x += n.x/length;
			n1.y += n.y/length;
			n1.z += n.z/length;

		}
		newVertsx[ivert] = v1.x/size + v0.x;
		newVertsy[ivert] = v1.y/size + v0.y;
		newVertsz[ivert] = v1.z/size + v0.z;

		float length1 = sqrt(n1.x*n1.x + n1.y*n1.y + n1.z*n1.z);
		newNormsx[ivert] = n1.x/length1;
		newNormsy[ivert] = n1.y/length1;
		newNormsz[ivert] = n1.z/length1;

	}



}


//---------------------------------------------------------
// Naive implementation. Slower in the best case, but without 
// any assumptions attached.

__global__ void cuda_mesh_smooth_naive(
	size_t nidxs, const size_t *vertidxs, 
	const size_t *vertadj, const size_t *vertadjOffsets, 
	const float* vertsx, const float* vertsy, const float* vertsz,
	const float* normsx, const float* normsy, const float* normsz,
	float* newVertsx, float* newVertsy, float* newVertsz,
	float* newNormsx, float* newNormsy, float* newNormsz)
{
	const size_t idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx < nidxs) {
		size_t ivert = vertidxs[idx];
		float3 v0, n0;
		v0.x = vertsx[ivert];
		v0.y = vertsy[ivert];
		v0.z = vertsz[ivert];
		n0.x = normsx[ivert];
		n0.y = normsy[ivert];
		n0.z = normsz[ivert];

		float3 v1 = make_float3(0.0f, 0.0f, 0.0f);
		float3 n1 = make_float3(0.0f, 0.0f, 0.0f);

		size_t offset = vertadjOffsets[idx];
		size_t offsetNxt = vertadjOffsets[idx+1];
		size_t size = offsetNxt - offset;
		for (size_t i = offset; i < offsetNxt; ++i) {
			size_t a = vertadj[i];
			float3 v, n;
			v.x = vertsx[a];
			v.y = vertsy[a];
			v.z = vertsz[a];
			n.x = normsx[a];
			n.y = normsy[a];
			n.z = normsz[a];

			v.x = v.x - v0.x;
			v.y = v.y - v0.y;
			v.z = v.z - v0.z;
			float vn0 = v.x*n0.x + v.y*n0.y + v.z*n0.z;
			float vn = v.x*n.x + v.y*n.y + v.z*n.z;
			v1.x += 0.5f*v.x + 0.25f*(vn*n.x - vn0*n0.x);
			v1.y += 0.5f*v.y + 0.25f*(vn*n.y - vn0*n0.y);
			v1.z += 0.5f*v.z + 0.25f*(vn*n.z - vn0*n0.z);

			n.x = 0.5f*(n.x + n0.x);
			n.y = 0.5f*(n.y + n0.y);
			n.z = 0.5f*(n.z + n0.z);
			float length = sqrt(n.x*n.x + n.y*n.y + n.z*n.z);
			n1.x += n.x/length;
			n1.y += n.y/length;
			n1.z += n.z/length;
		}
		newVertsx[ivert] = v1.x/size + v0.x;
		newVertsy[ivert] = v1.y/size + v0.y;
		newVertsz[ivert] = v1.z/size + v0.z;

		float length1 = sqrt(n1.x*n1.x + n1.y*n1.y + n1.z*n1.z);
		newNormsx[ivert] = n1.x/length1;
		newNormsy[ivert] = n1.y/length1;
		newNormsz[ivert] = n1.z/length1;

	}

}


//---------------------------------------------------------
// Wrapper calling the kernels

extern "C"  void cuda_mesh_smooth(
	int nSweeps, size_t nverts, size_t nidxs, size_t nadj, 
	unsigned int nthreadsPerBlock, bool localizedVerts,
	const size_t *vertidxs, const size_t *vertadj, 
	const size_t *vertadjOffsets,
	float *vertsx, float *vertsy, float *vertsz, 
	float *normsx, float *normsy, float *normsz, bool& success)
{        
	success = false;
	//hipEvent_t start, end;
	//float time = 0;

	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if ( deviceCount == 0 ) return;

	if (nSweeps <= 0 || nverts <= 0 || nidxs <= 0 || nadj <= 0) {
		success = true;
		return;
	}

	DevArray<size_t> d_vertidxs(nidxs);
	DevArray<size_t> d_vertadj(nadj);
	DevArray<size_t> d_vertadjOffsets(nidxs+1);
	DevArray<float> d_vertsx(nverts);
	DevArray<float> d_vertsy(nverts);
	DevArray<float> d_vertsz(nverts);
	DevArray<float> d_normsx(nverts);
	DevArray<float> d_normsy(nverts);
	DevArray<float> d_normsz(nverts);
	DevArray<float> d_newVertsx(nverts);
	DevArray<float> d_newVertsy(nverts);
	DevArray<float> d_newVertsz(nverts);
	DevArray<float> d_newNormsx(nverts);
	DevArray<float> d_newNormsy(nverts);
	DevArray<float> d_newNormsz(nverts);

	d_vertidxs.set(&vertidxs[0], nidxs);
	d_vertadj.set(&vertadj[0], nadj);
	d_vertadjOffsets.set(&vertadjOffsets[0], nidxs+1);
	d_vertsx.set(&vertsx[0], nverts);
	d_vertsy.set(&vertsy[0], nverts);
	d_vertsz.set(&vertsz[0], nverts);
	d_normsx.set(&normsx[0], nverts);
	d_normsy.set(&normsy[0], nverts);
	d_normsz.set(&normsz[0], nverts);

	d_newVertsx.set(&d_vertsx, nverts);
	d_newVertsy.set(&d_vertsy, nverts);
	d_newVertsz.set(&d_vertsz, nverts);
	d_newNormsx.set(&d_normsx, nverts);
	d_newNormsy.set(&d_normsy, nverts);
	d_newNormsz.set(&d_normsz, nverts);


	DevArray<float> *pd_vertsx = &d_vertsx;
	DevArray<float> *pd_vertsy = &d_vertsy;
	DevArray<float> *pd_vertsz = &d_vertsz;
	DevArray<float> *pd_normsx = &d_normsx;
	DevArray<float> *pd_normsy = &d_normsy;
	DevArray<float> *pd_normsz = &d_normsz;
	DevArray<float> *pd_newVertsx = &d_newVertsx;
	DevArray<float> *pd_newVertsy = &d_newVertsy;
	DevArray<float> *pd_newVertsz = &d_newVertsz;
	DevArray<float> *pd_newNormsx = &d_newNormsx;
	DevArray<float> *pd_newNormsy = &d_newNormsy;
	DevArray<float> *pd_newNormsz = &d_newNormsz;

	auto swap_vert_norm_buffers = [&]() ->void {
		std::swap(pd_vertsx, pd_newVertsx);
		std::swap(pd_vertsy, pd_newVertsy);
		std::swap(pd_vertsz, pd_newVertsz);
		std::swap(pd_normsx, pd_newNormsx);
		std::swap(pd_normsy, pd_newNormsy);
		std::swap(pd_normsz, pd_newNormsz);
	};

	size_t nblocks = (nidxs + nthreadsPerBlock - 1)/nthreadsPerBlock;
	unsigned int nshared = 2*nthreadsPerBlock*sizeof(float3);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_mesh_smooth_naive), hipFuncCachePreferL1);
//	hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_mesh_smooth_sharedmem), hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_mesh_smooth_sharedmem), hipFuncCachePreferEqual);

	//hipEventCreate(&start);
	//hipEventCreate(&end);
	//hipEventRecord(start);


	if (localizedVerts) {
		// Vertices ordered into patches on the surface
		for (int i = 0; i < nSweeps; ++i) {
			cuda_mesh_smooth_sharedmem<<<nblocks, nthreadsPerBlock, nshared>>>(
				nverts, nidxs, d_vertidxs.getData(), d_vertadj.getData(),
				d_vertadjOffsets.getData(),
				pd_vertsx->getData(), pd_vertsy->getData(), pd_vertsz->getData(),
				pd_normsx->getData(), pd_normsy->getData(), pd_normsz->getData(),
				pd_newVertsx->getData(), pd_newVertsy->getData(), pd_newVertsz->getData(),
				pd_newNormsx->getData(), pd_newNormsy->getData(), pd_newNormsz->getData());
			//Swap buffer pointers, new coordinates becoming current
			if (i < nSweeps-1) swap_vert_norm_buffers();
		}
	}
	else {
		for (int i = 0; i < nSweeps; ++i) {
			cuda_mesh_smooth_naive<<<nblocks, nthreadsPerBlock>>>(
				nidxs, d_vertidxs.getData(), d_vertadj.getData(),
				d_vertadjOffsets.getData(),
				pd_vertsx->getData(), pd_vertsy->getData(), pd_vertsz->getData(),
				pd_normsx->getData(), pd_normsy->getData(), pd_normsz->getData(),
				pd_newVertsx->getData(), pd_newVertsy->getData(), pd_newVertsz->getData(),
				pd_newNormsx->getData(), pd_newNormsy->getData(), pd_newNormsz->getData());
			//Swap buffer pointers, new coordinates becoming current
			if (i < nSweeps-1) swap_vert_norm_buffers();  
		}
	}

	hipDeviceSynchronize();
	//hipEventRecord(end);
	//hipEventSynchronize(end);
	//hipEventElapsedTime(&time, start, end);
	//cout << "Execution Time: " << time << endl;

	pd_newVertsx->get(&vertsx[0], nverts);
	pd_newVertsy->get(&vertsy[0], nverts);
	pd_newVertsz->get(&vertsz[0], nverts);
	pd_newNormsx->get(&normsx[0], nverts);
	pd_newNormsy->get(&normsy[0], nverts);
	pd_newNormsz->get(&normsz[0], nverts);

	success = true;

}
